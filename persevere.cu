#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "/content/drive/MyDrive/kate.h"

__global__ void bfs_kernel(int *d_graph_nodes, int *d_graph_edges, int *d_cost, int *d_graph_active, int *d_updating_graph_active, int k, int *d_count, int no_of_nodes) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;


    if (tid < no_of_nodes && d_graph_active[tid] != -1) {
        int node = d_graph_active[tid];
        for (int i = d_graph_nodes[node]; i < d_graph_nodes[node + 1]; i++) {
            int id = d_graph_edges[i];
            if (atomicMin(&d_cost[id], k) > k) {
                int pos = atomicAdd(d_count, 1);
                d_updating_graph_active[pos] = id;
            }
        }
    }
}
    
    
    
    


int main() {
    int no_of_nodes = ver;
    int edge_list_size = edg;
    int source = 0;

    h_graph_active[0] = source;
    
    int *d_graph_nodes, *d_graph_edges, *d_cost, *d_graph_active, *d_updating_graph_active, *d_count;
    hipMalloc((void**)&d_graph_nodes, sizeof(int) * (no_of_nodes + 1));
    hipMalloc((void**)&d_graph_edges, sizeof(int) * edge_list_size);
    hipMalloc((void**)&d_cost, sizeof(int) * no_of_nodes);
    hipMalloc((void**)&d_graph_active, sizeof(int) * no_of_nodes);
    hipMalloc((void**)&d_updating_graph_active, sizeof(int) * no_of_nodes);
    hipMalloc((void**)&d_count, sizeof(int));
    
    hipMemcpy(d_graph_nodes, h_graph_nodes, sizeof(int) * (no_of_nodes + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_graph_edges, h_graph_edges, sizeof(int) * edge_list_size, hipMemcpyHostToDevice);
    hipMemcpy(d_cost, h_cost, sizeof(int) * no_of_nodes, hipMemcpyHostToDevice);
    hipMemcpy(d_graph_active, h_graph_active, sizeof(int) * no_of_nodes, hipMemcpyHostToDevice);
 
    for (int i = 0; i < ver; i++) {
        h_cost[i] = -1;
    }
    
     
    h_cost[source] = 0;
    hipMemcpy(d_cost, h_cost, sizeof(int) * no_of_nodes, hipMemcpyHostToDevice);
    
    int count1 = 1;
    int k = 0;
    int *h_count = (int*)malloc(sizeof(int));
    int var = *h_count;
    
    
     do {
        k++;
        *h_count = 0;
        hipMemcpy(d_count, h_count, sizeof(int), hipMemcpyHostToDevice);
        
        int num_blocks = (count1 + 255) / 256;
        bfs_kernel<<<num_blocks, 256>>>(d_graph_nodes, d_graph_edges, d_cost, d_graph_active,    d_updating_graph_active, k, d_count, ver);

        hipMemcpy(h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);
        count1 = *h_count;

        hipMemcpy(d_graph_active, d_updating_graph_active, sizeof(int) * count1, hipMemcpyDeviceToDevice);

    } while (var);
    
    hipMemcpy(h_cost, d_cost, sizeof(int) * no_of_nodes, hipMemcpyDeviceToHost);

    hipFree(d_graph_nodes);
    hipFree(d_graph_edges);
    hipFree(d_cost);
    hipFree(d_graph_active);
    hipFree(d_updating_graph_active);
    hipFree(d_count);
    free(h_count);
 }
