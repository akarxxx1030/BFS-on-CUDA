#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void sample (int *d_array, int *d_sum, int N){
__shared__ s_array;
int tid = blockIdx.x * blockDim.x + threadIdx.x;
/*
if (tid < N)
s_array[i] = d_array[i];
__syncthreads();

if (tid < N){
s_array[tid] += tid;
atomicAdd(d_sum, s_array[tid]);
}
__syncthreads(); 
   
if (tid < N)
d_array[tid] = s_array[tid];   
}*/

if(tid<N){
d_array[tid]+=tid;
atomicAdd(d_sum,d_array[tid]);}
//d_sum +=d_array[tid];
}


int main(){
  int h_sum=0;
  int *d_sum;
int h_array[10]={1,2,3,4,5,6,7,8,9,10};
int block_size = 256;
int N = sizeof(h_array)/sizeof(int);
int *d_array;
hipMalloc((void**)&d_array, sizeof(int)*N);
hipMalloc((void**)&d_sum, sizeof(int));
hipMemcpy(d_array, h_array, sizeof(int)*N, hipMemcpyHostToDevice);
hipMemset(d_sum, 0, sizeof(int));
int num_blocks = (N+block_size-1)/block_size;
//sample <<< num_blocks, block_size>>>(d_array, d_sum, N);
sample <<<4,block_size>>>(d_array, d_sum, N);
hipMemcpy(h_array, d_array, sizeof(int) * N, hipMemcpyDeviceToHost);
hipMemcpy(&h_sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);
hipFree(d_array);
hipFree(d_sum);
for(int i=0;i<N;i++){
printf("%d\n",h_array[i]);}
printf("%d\n",h_sum);
return 0;
}


