#include<stdio.h>
#include<hip/hip_runtime.h>
#define L 16777216


__global__ void sample(double *d_array, double *d_sum, int N) {
    __shared__ double s_array[1024];
    __shared__ double d_tb_sum;

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;
    double local_sum=0.0;
    if(tid<N) {
    local_sum = d_array[tid]+tid;
    }
    s_array[local_tid]=local_sum;
    __syncthreads();

    for(int stride=blockDim.x/2; stride>0; stride/=2) {
        if (local_tid < stride) {
            s_array[local_tid]+= s_array[local_tid+stride];}
        __syncthreads();}

    if(local_tid == 0){
    atomicAdd(&d_tb_sum, s_array[0]);}
    __syncthreads();
    
    if(local_tid == 0){
        atomicAdd(d_sum, d_tb_sum);}
}


int main(){
  double h_sum=0.0;
  double *d_sum;
//int h_array[10]={1,2,3,4,5,6,7,8,9,10};
double h_array[L];
/*for(int i =0; i<L; i++)
h_array[i]=1;*/
int block_size = 1024;
int N = sizeof(h_array)/sizeof(int);
double *d_array;
hipMalloc(&d_array, sizeof(double)*L);
hipMalloc(&d_sum, sizeof(double));
hipMemcpy(d_array, h_array, sizeof(double)*L, hipMemcpyHostToDevice);
hipMemset(d_sum, 0, sizeof(double));
int num_blocks = (N+block_size-1)/block_size;
//sample <<< num_blocks, block_size>>>(d_array, d_sum, N);
sample <<<num_blocks,block_size>>>(d_array, d_sum, N);
hipDeviceSynchronize();
hipMemcpy(h_array, d_array, sizeof(double) * L, hipMemcpyDeviceToHost);
hipMemcpy(&h_sum, d_sum, sizeof(double), hipMemcpyDeviceToHost);
hipFree(d_array);
hipFree(d_sum);
/*for(int i=0;i<N;i++){
printf("%d\n",h_array[i]);}*/
printf("%lf\n",h_sum);
return 0;
}
